
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#define THDS_PER_BLK 256
#define N 1000000
__global__ void prime_calc(int* cnt_arr_d)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;
	int i;
	int flag = 0;
	cnt_arr_d[myid] = 0;
	if((myid > 2) && (myid < N))
	{
		for(i=2; i<myid;i++)
		{
			if((myid%i) == 0)
			{
			    flag = 1;
			    break;
			}	
		}
	        if(flag == 0)
        	{
        		cnt_arr_d[myid] = 1;
        	}
	}
}
int main()
{
	int *cnt_arr_d;
	int cnt_arr[N];
	int i=0;
	int total = 0;
	double exe_time;
	struct timeval stop_time, start_time;
    	int thds_per_block = THDS_PER_BLK;
	int num_blocks = (N/thds_per_block)+1;
	
	gettimeofday(&start_time, NULL);
	
	hipMalloc(&cnt_arr_d, N*sizeof(int));

	prime_calc<<< num_blocks,thds_per_block >>>(cnt_arr_d); 

	hipMemcpy(cnt_arr, cnt_arr_d, N*sizeof(int), hipMemcpyDeviceToHost);

	for(i=0; i<N; i++)
	{
		total += cnt_arr[i];
	}
	
    	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
    	printf("\n Total = %d and exe_time = %lf\n", total+1, exe_time);	
	hipFree(cnt_arr_d);
	
}
