
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void arradd(int* md, int* nd, int* pd)
{
	int myid = threadIdx.x;
	
	pd[myid] = md[myid] + nd[myid];
}


int main()
{
	int N = 1024;
	int size = N * sizeof(int);
	int m[N], n[N], p[N],*md, *nd,*pd;
	int i=0;

	for(i=0; i<N; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}

	hipMalloc(&md, size);
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);

	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);

	hipMalloc(&pd, size);

	dim3   DimGrid(1, 1);     
	dim3   DimBlock(N, 1);   


	arradd<<< DimGrid,DimBlock >>>(md,nd,pd);

	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);

//	printf("Task Done...\n");
	for(i=0; i<N; i++ )
	{
		if(p[i] != (i+i)){
			printf("Failed at %d.\n",i);
			break;
		}
	}
		
	hipFree(md); 
	hipFree(nd);
	hipFree(pd);	
}

