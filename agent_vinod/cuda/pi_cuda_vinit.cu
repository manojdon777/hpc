
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void PI( double steps, double* sum_arr_d)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;	
        double x, sum=0.0;
        x = (myid)*steps;
        sum = sum + 4.0/(1.0+x*x);
        sum_arr_d[myid] = sum;
        
}


int main()
{
        static long num_step = 1000000;
	double sum_arr[1000000], *sum_arr_d;
	double steps;
	int i; 
	double total=0.0, pi;
	
	int size = 1000000 * sizeof(double);
	steps = 1.0/(double)num_step;
	
	hipMalloc(&sum_arr_d, size);
	
	dim3   DimGrid(1000, 1);     
	dim3   DimBlock(1000, 1);   
	
	PI<<< DimGrid,DimBlock >>>(steps, sum_arr_d);
	
	hipMemcpy(sum_arr, sum_arr_d, size, hipMemcpyDeviceToHost);

	for(i=0; i<1000000; i++ )
	{
		total+=sum_arr[i];
	}
	pi = steps * total;
	printf("PI = %lf ,  sum = %lf\n",pi, total);
	hipFree(sum_arr_d);	
}
