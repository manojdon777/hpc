
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 32
__global__ void mat_vector_mul(int *A_d, int *B_d, int *C_d)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;
	int blk_id = blockIdx.x;
	C_d[blk_id] += A_d[myid] * B_d[blk_id];
}

int main()
{
	int size = N * sizeof(int);
	int A[N][N], B[N], C[N], *A_d, *B_d, *C_d;
	int i, j;
	
        for(i=0; i<N; i++)
        {
        	B[i] = 1;
                for(j=0; j<N; j++)
                {
                        A[i][j] = 1;
                }
        }
        
        hipMalloc(&A_d, N*N*sizeof(int));
        hipMalloc(&B_d, size);
        hipMalloc(&C_d, size);
        
        hipMemcpy(A_d, A, N*size, hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
        
        dim3 blocks(32,1);
        dim3 threads(32,1);
        
        mat_vector_mul<<< blocks, threads >>>(A_d, B_d, C_d);
        
        hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
        
        for(i=0; i<N; i++)
        {
        	printf("\t%d", C[i]);
        }
        
        hipFree(A_d);
        hipFree(B_d);
        hipFree(C_d);
}



